#include "hip/hip_runtime.h"
__global__ void addKernel(int *c,  int *a,  int *b, unsigned int size)
{
	// ��� ������� ����
int i = blockIdx.x *blockDim.x + threadIdx.x;
c[i] = a[i] + b[i];
}

#define kernel addKernel
#include "mainGPU.h"

